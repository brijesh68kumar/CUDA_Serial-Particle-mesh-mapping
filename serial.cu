
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "time.h"

int main(int argc, char *argv[])
{

        int max = 64, i,j,lp=1;
        int top,bottom,left,right;

        float net[64][64];
        float x,y, fL, fR, fB, fT;

        unsigned int par=850000,loop=1000;

        struct timespec start,stop;
        double t1=0,t2=0,result=0;


        for (i=0;i<max;i++)
                for (j=0; j<max;j++)
                        net[i][j]=0;

 //------------------calculate Starting time----------------------
        clock_gettime(CLOCK_REALTIME,&start);
        t1 = start.tv_sec + (start.tv_nsec/pow(10,9));

	for(lp;lp<loop;lp++){
        for ( i = 0; i < par; ++i)
        {
                x = ((float)rand()/(float)(RAND_MAX) * (float)max);
                y = ((float)rand()/(float)(RAND_MAX) * (float)max);

                left = (int)floor(x);
                right = left + 1;

                bottom = (int)floor(y);
                top = bottom +1;

        if (top>=max||bottom>=max||left>=max||right>=max)
        {
            continue;
        }

                fL = x - left;
                fR = 1 - fL;

                fB = y - bottom;
                fT = 1 - fB;

                net[left][bottom]       =       net[left][bottom]       +( fT * fR ) ;
                net[right][bottom]      =       net[right][bottom]      +( fT * fL ) ;
                net[left][top]          =       net[left][top]          +( fB * fR ) ;
                net[right][top]         =       net[right][top]         +( fB * fL ) ;
        }
	}


 //---------------calculate End time-------------------------
        clock_gettime(CLOCK_REALTIME,&stop);
        t2 = stop.tv_sec + (stop.tv_nsec/pow(10,9));



        FILE *f = fopen("file2.txt", "w");
        if (f == NULL)
        {
        printf("Error opening file!\n");
        exit(1);
        }

	float avg= par/(max*max);

                for ( i = 0; i < max; ++i)
                {
                        for ( j = 0; j < max; j++)
                        {
//                                printf ("%f ,",net[i][j] );
                                fprintf (f,"%f,",((net[i][j])/avg) );
                        }
//                       printf ("\n" );
                        fprintf (f,"\n" );
                }
        fclose(f);

        result = t2 - t1 ;
        printf("its done:\t%lf s\n", result);


        return 0;
}

